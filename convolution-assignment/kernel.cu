#include "hip/hip_runtime.h"
/******************************************************************************
 *cr
 *cr            (C) Copyright 2010 The Board of Trustees of the
 *cr                        University of Illinois
 *cr                         All Rights Reserved
 *cr
 ******************************************************************************/

__constant__ float M_c[FILTER_SIZE][FILTER_SIZE];

__global__ void convolution(Matrix N, Matrix P)
{
	/********************************************************************
	Determine input and output indexes of each thread
	Load a tile of the input image to shared memory
	Apply the filter on the input image tile
	Write the compute values to the output image at the correct indexes
	********************************************************************/
     
     __shared__ float N_s[BLOCK_SIZE][BLOCK_SIZE];
       
     
     int tx = threadIdx.x;
     int ty = threadIdx.y;

     int row_o = ty + blockIdx.y * TILE_SIZE;
     int col_o = blockIdx.x * TILE_SIZE + tx;

     int row_i = row_o - (FILTER_SIZE/2.0);
     int col_i = col_o - (FILTER_SIZE/2.0);


    //Check to see if mask hangs off, if so set to 0 
    if(row_i >= 0 && row_i < N.height && col_i >= 0 && col_i < N.width){
      N_s[ty][tx] = N.elements[row_i * N.width + col_i];
    }else{
      N_s[ty][tx] = 0.0f;
    }
    
    //Wait for threads to sync
    __syncthreads();
    
    float temp = 0.0f;
    if(tx < TILE_SIZE && ty < TILE_SIZE){
        
        for(int i = 0; i < FILTER_SIZE; i++)
           for(int j = 0; j < FILTER_SIZE; j++)
               temp += M_c[i][j] * N_s[i + ty][j + tx];
           
           
       if((row_o < P.height) && (col_o < P.width)){
             P.elements[row_o * P.width + col_o] = temp;
         }
      
    }    
}      
